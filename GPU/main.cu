#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <vector>
#include <cmath>
#include <string>
#include <random>
#include <omp.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
#include <cstring>
using namespace std;

struct tipo_salto{

    int salto;
};


struct seq{

    string sub_a;
    string sub_b;
    int score;

};



int main(){

    int N, M;
    cin >> N;
    cin >> M;

    vector<char> a;
    vector<char> b;
 
    vector<vector<char>> subseq_a;
    vector<vector<char>> subseq_b;
 
  
    a.reserve(N+1);
    b.reserve(M+1);
    subseq_a.reserve(N+1);
    subseq_b.reserve(M+1);
   
  
    for(int i=0; i<N; i++){

        cin>> a[i];
       
    }for(int i=0; i<M; i++){

        cin>> b[i];
        
    }


    for(int i=0; i< N; i++){
        string s="-";
        vector<char> w;
        for(int j=i; j<N; j++){

            s+=a[j];
            w.push_back(a[j]);
            
            // subseq_a.push_back(s);
        }
        subseq_a.push_back(w);
    }

    int suba_size= subseq_a.size();

    for(int i=0; i< M; i++){
        string s="-";
        vector<char> w;
        for(int j=i; j<M; j++){

            s+=b[j];
            // subseq_b.push_back(s);
        }
        subseq_b.push_back(w);

    }

    int subb_size= subseq_b.size();

    thrust::device_vector<vector<char>> sub_a(suba_size); 
    thrust::device_vector<vector<char>> sub_b(subb_size); 
    thrust::device_vector<seq> comb(subb_size* suba_size); 


    thrust::fill(subseq_a.begin(), subseq_a.end(), sub_a.begin()); 
    thrust::fill(subseq_b.begin(), subseq_b.end(), sub_b.begin()); 





    return 0;
}

// int main()
// {
//     int N, M;
//     cin >> N;
//     cin >> M;

//     vector<char> a;
//     vector<char> b;
 
//     vector<string>subseq_a;
//     vector<string>subseq_b;

//     vector<seq> combinacoes;
//     vector<vector<int>> H(N+1, vector<int>(M+1,0));


    
  
//     a.reserve(N+1);
//     b.reserve(M+1);
//     subseq_a.reserve(N+1);
//     subseq_b.reserve(M+1);
   
  
//     for(int i=0; i<N; i++){

//         cin>> a[i];
       
//     }for(int i=0; i<M; i++){

//         cin>> b[i];
        
//     }


//     for(int i=0; i< N; i++){
//         string s="-";
//         for(int j=i; j<N; j++){

//             s+=a[j];
//             subseq_a.push_back(s);
//         }
//     }

//     int suba_size= subseq_a.size();

//     for(int i=0; i< M; i++){
//         string s="-";
//         for(int j=i; j<M; j++){

//             s+=b[j];
//             subseq_b.push_back(s);
//         }
//     }

//     int subb_size= subseq_b.size();

//     string best_a;
//     string best_b;
//     int max_Score=0;

//     // Gerar todas as combinacoes de subsequencias em um vetor structs(a,b,score) 
//     // Percorrer com parallel for  e calcular o score
//     // Varrer o vetor sem openmp pra pegar o maior score
  
//     for(int i=0; i<subb_size; i++){
//         for(int j=0; j<suba_size; j++){

//             vector<char> sa(subseq_a[j].begin(), subseq_a[j].end());
//             vector<char> sb(subseq_b[i].begin(), subseq_b[i].end());
//             seq subs;
//             subs.sub_a= sa;
//             subs.sub_b=sb;
//             subs.score=0;

//             combinacoes.push_back(subs);
           
//         }    
//     }

//     int maxScore=0;
  
//     #pragma omp parallel for  firstprivate(H)
//     for( int i=0; i< int(combinacoes.size()); i++){

//         for (int linha=1; linha < int(combinacoes[i].sub_a.size()); linha++){
        
//                 for(int coluna=1; coluna<int(combinacoes[i].sub_b.size()); coluna++){
                
//                     int score;
//                     int diagonal; //0
//                     int delecao; //1
//                     int insercao; //2
//                     score=(combinacoes[i].sub_a[linha]==combinacoes[i].sub_b[coluna])?2:-1;
//                     diagonal= H[linha-1][coluna-1]+ score;
//                     delecao=H[linha-1][coluna]-1;
//                     insercao=H[linha][coluna-1]-1;

//                     H[linha][coluna]= std::max({0, diagonal, delecao, insercao});

//                 }
            


//         }
        

//         for (int b = 0; b < int(combinacoes[i].sub_a.size()); b++) {
//             for (int n = 0; n < int(combinacoes[i].sub_b.size()); n++) {       
//                 if (H[b][n] > maxScore) {
//                     maxScore=H[b][n];
//                 }
//             }
//         }

//         combinacoes[i].score=maxScore;
//     }

//     for(int i=0; i<int(combinacoes.size()); i++){

      
//         if(combinacoes[i].score> max_Score){
//             max_Score=combinacoes[i].score;
//             // best_a="";
//             // best_b="";

//             // for(int j=0; j< int(combinacoes[i].sub_a.size()); j++){

//             //     best_a+=combinacoes[i].sub_a[j];

//             // }
//             // for(int j=0; j< int(combinacoes[i].sub_b.size()); j++){

//             //     best_b+=combinacoes[i].sub_b[j];

//             // }

            

//         }
//     }


//     cout<<best_a<< endl;
//     cout<< best_b<<endl;
//     cout<< max_Score<<endl;


    
        
    
  
 
            

// return 0;
// }